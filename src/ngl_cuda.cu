#include "hip/hip_runtime.h"
#include "ngl_cuda.cuh"
#include <cstdio>
#include <vector>
#include <map>
#include <algorithm>

#define cudaErrchk(ans) { GPUAssert((ans), __FILE__, __LINE__); }
inline void GPUAssert(hipError_t code, const char *file, int line,
                      bool abort=true) {
	if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file,
                line);
		if (abort) exit(code);
	}
}

namespace nglcu {
    dim3 block_size(32, 32);
    dim3 grid_size(4, 4);
    dim3 block_size_1D(1024);
    dim3 grid_size_1D(16);

    __global__
    void prune_discrete_d(float *X, int *edgesIn, const int N, const int D,
                        const int K, float *erTemplate, const int steps,
                        int *edgesOut) {
        int index_x = blockIdx.x * blockDim.x + threadIdx.x;
        int stride_x = blockDim.x * gridDim.x;

        int index_y = blockIdx.y * blockDim.y + threadIdx.y;
        int stride_y = blockDim.y * gridDim.y;

        // References to points in X
        float *p, *q, *r;

        //TODO: Fix this to be dynamically allocated
        // Computed vectors representing the edge under test pq and the vector from
        // one end point to a third point r (We will iterate over all possible r's)
        float pq[10] = {};
        float pr[10] = {};

        // Different iterator/indexing variables i, j, and n are rows in X
        // representing p, q, and r, respectively
        // k is the nearest neighbor, d is the dimension
        int i, j, k, k2, d, n;

        // t is the parameterization of the projection of pr onto pq
        // In layman's terms, this is the length of the shadow pr casts onto pq
        // lookup is the
        float t;
        int lookup;

        // Some other temporary variables
        float length_squared;
        float squared_distance_to_edge;
        float minimum_allowable_distance;

        for (k = index_y; k < K; k += stride_y) {
            for (i = index_x; i < N; i += stride_x) {

                p = &(X[D*i]);
                j = edgesIn[K*i+k];
                q = &(X[D*j]);

                length_squared = 0;
                for(d = 0; d < D; d++) {
                    pq[d] = p[d] - q[d];
                    length_squared += pq[d]*pq[d];
                }
                // A point should not be connected to itself
                if(length_squared == 0) {
                    edgesOut[K*i+k] = -1;
                    continue;
                }

                // for(n = 0; n < N; n++) {
                for(k2 = 0; k2 < 2*K; k2++) {
                    n = (k2 < K) ? edgesIn[K*i+k2] : edgesIn[K*j+(k2-K)];
                    r = &(X[D*n]);

                    t = 0;
                    for(d = 0; d < D; d++) {
                        pr[d] = p[d] - r[d];
                        t += pr[d]*pq[d];
                    }

                    t /= length_squared;
                    lookup = __float2int_rd(abs(steps * (2 * t - 1))+0.5);
                    if (lookup >= 0 && lookup <= steps) {
                        squared_distance_to_edge = 0;
                        for(d = 0; d < D; d++) {
                            squared_distance_to_edge += (pr[d] - pq[d]*t)*(pr[d] - pq[d]*t);
                        }
                        minimum_allowable_distance = sqrt(length_squared)*erTemplate[lookup];

                        if(sqrt(squared_distance_to_edge) < minimum_allowable_distance) {
                            edgesOut[K*i+k] = -1;
                            break;
                        }
                    }
                }
            }
        }
    }

    __global__
    void prune_discrete_relaxed_d(float *X, int *edgesIn, const int N,
                                  const int D, const int K, float *erTemplate,
                                  const int steps, int *edgesOut) {
        int index_x = blockIdx.x * blockDim.x + threadIdx.x;
        int stride_x = blockDim.x * gridDim.x;

        // References to points in X
        float *p, *q, *r;

        //TODO: Fix this to be dynamically allocated
        // Computed vectors representing the edge under test pq and the vector from
        // one end point to a third point r (We will iterate over all possible r's)
        float pq[10] = {};
        float pr[10] = {};

        // Different iterator/indexing variables i, j, and n are rows in X
        // representing p, q, and r, respectively
        // k is the nearest neighbor, d is the dimension
        int i, j, k, k2, d, n;

        // t is the parameterization of the projection of pr onto pq
        // In layman's terms, this is the length of the shadow pr casts onto pq
        // lookup is the
        float t;
        int lookup;

        // Some other temporary variables
        float length_squared;
        float squared_distance_to_edge;
        float minimum_allowable_distance;

        for (k = 0; k < K; k++) {
            for (i = index_x; i < N; i += stride_x) {

                p = &(X[D*i]);
                j = edgesIn[K*i+k];
                q = &(X[D*j]);

                length_squared = 0;
                for(d = 0; d < D; d++) {
                    pq[d] = p[d] - q[d];
                    length_squared += pq[d]*pq[d];
                }
                // A point should not be connected to itself
                if(length_squared == 0) {
                    edgesOut[K*i+k] = -1;
                    continue;
                }

                // This loop presumes that all nearer neighbors have
                // already been processed
                for(k2 = 0; k2 < k; k2++) {
                    n = edgesOut[K*i+k2];
                    if (n == -1){
                        continue;
                    }
                    r = &(X[D*n]);

                    t = 0;
                    for(d = 0; d < D; d++) {
                        pr[d] = p[d] - r[d];
                        t += pr[d]*pq[d];
                    }

                    t /= length_squared;
                    lookup = __float2int_rd(abs(steps * (2 * t - 1))+0.5);
                    if (lookup >= 0 && lookup <= steps) {
                        squared_distance_to_edge = 0;
                        for(d = 0; d < D; d++) {
                            squared_distance_to_edge += (pr[d] - pq[d]*t)*(pr[d] - pq[d]*t);
                        }
                        minimum_allowable_distance = sqrt(length_squared)*erTemplate[lookup];

                        if(sqrt(squared_distance_to_edge) < minimum_allowable_distance) {
                            edgesOut[K*i+k] = -1;
                            break;
                        }
                    }
                }
            }
        }
    }

    __global__
    void prune_d(float *X, int *edgesIn, const int N, const int D, const int K,
                float lp, float beta, int *edgesOut) {
        int index_x = blockIdx.x * blockDim.x + threadIdx.x;
        int stride_x = blockDim.x * gridDim.x;

        int index_y = blockIdx.y * blockDim.y + threadIdx.y;
        int stride_y = blockDim.y * gridDim.y;

        float *p, *q, *r;

        float pq[10] = {};
        float pr[10] = {};

        int i, j, k, k2, d, n;
        float t;

        float length_squared;
        float squared_distance_to_edge;
        float minimum_allowable_distance;

        ////////////////////////////////////////////////////////////
        float xC, yC, radius, y;
        ////////////////////////////////////////////////////////////

        for (k = index_y; k < K; k += stride_y) {
            for (i = index_x; i < N; i += stride_x) {
                p = &(X[D*i]);
                j = edgesIn[K*i+k];
                q = &(X[D*j]);

                length_squared = 0;
                for(d = 0; d < D; d++) {
                    pq[d] = p[d] - q[d];
                    length_squared += pq[d]*pq[d];
                }
                // A point should not be connected to itself
                if(length_squared == 0) {
                    edgesOut[K*i+k] = -1;
                    continue;
                }

                // for(n = 0; n < N; n++) {
                for(k2 = 0; k2 < 2*K; k2++) {
                    n = (k2 < K) ? edgesIn[K*i+k2] : edgesIn[K*j+(k2-K)];
                    r = &(X[D*n]);

                    // t is the parameterization of the projection of pr onto pq
                    // In layman's terms, this is the length of the shadow pr casts onto pq
                    t = 0;
                    for(d = 0; d < D; d++) {
                        pr[d] = p[d] - r[d];
                        t += pr[d]*pq[d];
                    }

                    t /= length_squared;

                    if (t > 0 && t < 1) {
                        squared_distance_to_edge = 0;
                        for(d = 0; d < D; d++) {
                            squared_distance_to_edge += (pr[d] - pq[d]*t)*(pr[d] - pq[d]*t);
                        }

                        ////////////////////////////////////////////////////////////
                        // ported from python function, can possibly be improved
                        // in terms of performance
                        xC = 0;
                        yC = 0;

                        if (beta <= 1) {
                            radius = 1. / beta;
                            yC = powf(powf(radius, lp) - 1, 1. / lp);
                        }
                        else {
                            radius = beta;
                            xC = 1. - beta;
                        }
                        t = fabs(2*t-1);
                        y = powf(powf(radius, lp) - powf(t-xC, lp), 1. / lp) - yC;
                        minimum_allowable_distance = 0.5*y*sqrt(length_squared);

                        //////////////////////////////////////////////////////////
                        if(sqrt(squared_distance_to_edge) < minimum_allowable_distance) {
                            edgesOut[K*i+k] = -1;
                            break;
                        }
                    }
                }
            }
        }
    }

    __global__
    void prune_relaxed_d(float *X, int *edgesIn, const int N, const int D,
                         const int K, float lp, float beta, int *edgesOut) {
        int index_x = blockIdx.x * blockDim.x + threadIdx.x;
        int stride_x = blockDim.x * gridDim.x;

        // We should use a 1D structure for this since we need to guarantee
        // that other points have already been processed
        // int index_y = blockIdx.y * blockDim.y + threadIdx.y;
        // int stride_y = blockDim.y * gridDim.y;

        float *p, *q, *r;

        float pq[10] = {};
        float pr[10] = {};

        int i, j, k, k2, d, n;
        float t;

        float length_squared;
        float squared_distance_to_edge;
        float minimum_allowable_distance;

        ////////////////////////////////////////////////////////////
        float xC, yC, radius, y;
        ////////////////////////////////////////////////////////////

        for (i = index_x; i < N; i += stride_x) {
            for (k = 0; k < K; k++) {
                p = &(X[D*i]);
                j = edgesIn[K*i+k];
                q = &(X[D*j]);

                length_squared = 0;
                for(d = 0; d < D; d++) {
                    pq[d] = p[d] - q[d];
                    length_squared += pq[d]*pq[d];
                }
                // A point should not be connected to itself
                if(length_squared == 0) {
                    edgesOut[K*i+k] = -1;
                    continue;
                }

                // This loop presumes that all nearer neighbors have
                // already been processed
                for(k2 = 0; k2 < k; k2++) {
                    n = edgesOut[K*i+k2];
                    if (n == -1){
                        continue;
                    }
                    r = &(X[D*n]);

                    // t is the parameterization of the projection of pr onto pq
                    // In layman's terms, this is the length of the shadow pr casts onto pq
                    t = 0;
                    for(d = 0; d < D; d++) {
                        pr[d] = p[d] - r[d];
                        t += pr[d]*pq[d];
                    }

                    t /= length_squared;

                    if (t > 0 && t < 1) {
                        squared_distance_to_edge = 0;
                        for(d = 0; d < D; d++) {
                            squared_distance_to_edge += (pr[d] - pq[d]*t)*(pr[d] - pq[d]*t);
                        }

                        ////////////////////////////////////////////////////////////
                        // ported from python function, can possibly be improved
                        // in terms of performance
                        xC = 0;
                        yC = 0;

                        if (beta <= 1) {
                            radius = 1. / beta;
                            yC = powf(powf(radius, lp) - 1, 1. / lp);
                        }
                        else {
                            radius = beta;
                            xC = 1. - beta;
                        }
                        t = fabs(2*t-1);
                        y = powf(powf(radius, lp) - powf(t-xC, lp), 1. / lp) - yC;
                        minimum_allowable_distance = 0.5*y*sqrt(length_squared);

                        //////////////////////////////////////////////////////////
                        if(sqrt(squared_distance_to_edge) < minimum_allowable_distance) {
                            edgesOut[K*i+k] = -1;
                            break;
                        }
                    }
                }
            }
        }
    }

    float min_distance_from_edge(float t, float beta, float p) {
        float xC = 0;
        float yC = 0;
        float r;

        if (t > 1) {
            return 0;
        }
        if (beta <= 1) {
            r = 1. / beta;
            yC = powf(powf(r, p) - 1, 1. / p);
        }
        else {
            r = beta;
            xC = 1 - beta;
        }
        float y = powf(powf(r, p) - powf(t-xC, p), 1. / p) - yC;
        return 0.5*y;
    }

    void create_template(float * data, float beta, int p, int steps) {
        if (p < 0) {
            if (beta >= 1) {
                for (int i = 0; i <= steps; i++) {
                    data[i] = beta / 2.;
                }
            }
            else {
                for (int i = 0; i <= steps; i++) {
                    data[i] = 0.;
                }
            }
        }
        else {
            for (int i = 0; i <= steps; i++) {
                data[i] = min_distance_from_edge(float(i)/steps, beta, p);
            }
        }
    }

    void prune_discrete(float *X, int *edges, const int N, const int D,
                        const int K, float *erTemplate, const int steps,
                        const bool relaxed, const float beta, const float p) {
        float *x_d;
        int *edgesIn_d;
        int *edgesOut_d;
        float *erTemplate_d;
        cudaErrchk(hipMallocManaged(&x_d, N*D*sizeof(float)));
        cudaErrchk(hipMallocManaged(&edgesIn_d, N*K*sizeof(int)));
        cudaErrchk(hipMallocManaged(&edgesOut_d, N*K*sizeof(int)));
        cudaErrchk(hipMallocManaged(&erTemplate_d, (steps)*sizeof(float)));

        memcpy(x_d, X, N*D*sizeof(float));
        memcpy(edgesIn_d, edges, N*K*sizeof(float));
        memcpy(edgesOut_d, edges, N*K*sizeof(float));

        if (erTemplate != NULL) {
            memcpy(erTemplate_d, erTemplate, (steps)*sizeof(float));
        }
        else {
            float temp_erTemplate[steps];
            create_template(temp_erTemplate, beta, p, steps);
            memcpy(erTemplate_d, temp_erTemplate, (steps)*sizeof(float));
        }

        if (relaxed) {
            prune_discrete_relaxed_d<<<grid_size_1D, block_size_1D>>>(x_d,
                                                                      edgesIn_d,
                                                                      N,
                                                                      D,
                                                                      K,
                                                                      erTemplate_d,
                                                                      steps,
                                                                      edgesOut_d);
        }
        else {
            prune_discrete_d<<<grid_size, block_size>>>(x_d,
                                                        edgesIn_d,
                                                        N,
                                                        D,
                                                        K,
                                                        erTemplate_d,
                                                        steps,
                                                        edgesOut_d);
        }
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
            printf("Error: %s\n", hipGetErrorString(err));
        hipDeviceSynchronize();

        memcpy(edges, edgesOut_d, N*K*sizeof(float));

        hipFree(x_d);
        hipFree(edgesIn_d);
        hipFree(edgesOut_d);
        hipFree(erTemplate_d);
    }

    void prune(float *X, int *edges, const int N, const int D, const int K,
               bool relaxed, float beta, float lp) {
        float *x_d;
        int *edgesIn_d;
        int *edgesOut_d;
        hipMallocManaged(&x_d, N*D*sizeof(float));
        hipMallocManaged(&edgesIn_d, N*K*sizeof(int));
        hipMallocManaged(&edgesOut_d, N*K*sizeof(int));

        memcpy(x_d, X, N*D*sizeof(float));
        memcpy(edgesIn_d, edges, N*K*sizeof(float));
        memcpy(edgesOut_d, edges, N*K*sizeof(float));

        if (relaxed) {
            prune_relaxed_d<<<grid_size_1D, block_size_1D>>>(x_d, edgesIn_d, N, D, K, lp, beta, edgesOut_d);
        }
        else {
            prune_d<<<grid_size, block_size>>>(x_d, edgesIn_d, N, D, K, lp, beta, edgesOut_d);
        }

        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
            printf("Error: %s\n", hipGetErrorString(err));
        hipDeviceSynchronize();

        memcpy(edges, edgesOut_d, N*K*sizeof(float));

        hipFree(x_d);
        hipFree(edgesIn_d);
        hipFree(edgesOut_d);
    }

    vector_edge get_edge_list(int *edges, const int N, const int K) {
        int i, k;
        vector_edge edge_list;
        for(i = 0; i < N; i++) {
            for(k = 0; k < K; k++) {
                if (edges[i*K+k] != -1) {
                    edge_list.push_back(std::make_pair(i, edges[i*K+k]));
                }
            }
        }
        return edge_list;
    }

    void print_cuda_info() {
        struct hipDeviceProp_t properties;
        hipGetDeviceProperties(&properties, 0);
        fprintf(stderr, "using %d multiprocessors\n", properties.multiProcessorCount);
        fprintf(stderr, "max threads per processor: %d\n", properties.maxThreadsPerMultiProcessor);
        fprintf(stderr, "Grid Size: %dx%d\n", grid_size.x, grid_size.y);
        fprintf(stderr, "Block Size: %dx%d\n", block_size.x, block_size.y);
    }
}
